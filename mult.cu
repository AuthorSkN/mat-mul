
#include <hip/hip_runtime.h>
#include "iostream"
#include <fstream>
#include <string>

#define N	1024        // size of matrix is N*N
#define BLOCK_SIZE  16

__global__ void gpuMult ( float * a, float * b, int n, float * c )
{
    int   bx  = blockIdx.x;     
    int   by  = blockIdx.y;
    int   tx  = threadIdx.x;        
    int   ty  = threadIdx.y;
    float result = 0.0f;          
    int   idxA  = n * BLOCK_SIZE * by + n * ty;  
    int   idxB  = BLOCK_SIZE * bx + tx;

    for ( int idx = 0; idx < n; idx++ )
        result += a [idxA + idx] * b [idxB + idx*n];

    int idxC = n * BLOCK_SIZE * by + BLOCK_SIZE * bx;
    c [idxC + n * ty + tx] = result;
}

void cpuMult( float * a, float * b, int n, float * c ) {
	for (int rowIdxC = 0; rowIdxC < n; rowIdxC++) {
		for (int colIdxC = 0; colIdxC < n; colIdxC++) {
			float resultC = 0.0f;
			for (int idx = 0; idx < n; idx++) {
				int idxA = (rowIdxC * n) + idx;
				int idxB = colIdxC + (idx * n);
				resultC += a[idxA] * b[idxB];
			}
			int idxC = rowIdxC * n + colIdxC;
			c[idxC] = resultC;
		}
	}
}

int main() {
	float * a = new float [N*N];
    float * b = new float [N*N];
    float * c = new float [N*N];

    for ( int i = 0; i < N; i++ ) {
        for ( int j = 0; j < N; j++ ) {
            int k = N * i + j;

            a [k] = k;
            b [k] = k;
        }
	}
	
	//CPU-------------------------------------
	clock_t start_s = clock();
	cpuMult(a, b, N, c);
	clock_t stop_s = clock();
	std::cout << "Time CPU: " << (stop_s - start_s) / double(CLOCKS_PER_SEC) * 1000 <<  " ms\n";
	
	//GPU-------------------------------------
	int size = N * N * sizeof(float);
	
    float * adev = NULL;
    float * bdev = NULL;
    float * cdev = NULL;
    hipMalloc((void**)&adev, size);
    hipMalloc((void**)&bdev, size);
    hipMalloc((void**)&cdev, size);

    dim3 threads(BLOCK_SIZE, BLOCK_SIZE);
    dim3 blocks(N / threads.x, N / threads.y);

    hipEvent_t start, stop;
	hipEventCreate(&start);
    hipEventCreate(&stop);
    float gpuTime = 0.0f;

    hipEventRecord(start, 0);
    hipMemcpy(adev, a, size, hipMemcpyHostToDevice);
    hipMemcpy(bdev, b, size, hipMemcpyHostToDevice);

    gpuMult<<<blocks, threads>>>(adev, bdev, N, cdev);

    hipMemcpy(c, cdev, size, hipMemcpyDeviceToHost);
    hipEventRecord( stop, 0);

    hipEventSynchronize(stop);
    hipEventElapsedTime(&gpuTime, start, stop);
	std::cout << "Time GPU: " << gpuTime << " ms\n";
	
	hipEventDestroy(start);
    hipEventDestroy(stop);
    hipFree(adev);
    hipFree(bdev);
    hipFree(cdev);
    delete a;
    delete b;
    delete c;
	
	return 0;
}