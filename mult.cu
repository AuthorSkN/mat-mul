
#include <hip/hip_runtime.h>
#include "iostream"
#include <fstream>
#include <string>

#define N	4        // size of matrix is N*N

void cpuMult( float * a, float * b, int n, float * c ) {
	for (int rowIdxC = 0; rowIdxC < n; rowIdxC++) {
		for (int colIdxC = 0; colIdxC < n; colIdxC++) {
			float resultC = 0.0f;
			for (int idx = 0; idx < n; idx++) {
				int idxA = (rowIdxC * n) + idx;
				int idxB = colIdxC + (idx * n);
				resultC += a[idxA] * b[idxB];
			}
			int idxC = rowIdxC * n + colIdxC;
			c[idxC] = resultC;
		}
	}
}

int main() {
	float * a = new float [N*N];
    float * b = new float [N*N];
    float * c = new float [N*N];

    for ( int i = 0; i < N; i++ ) {
        for ( int j = 0; j < N; j++ ) {
            int k = N * i + j;

            a [k] = k;
            b [k] = k;
        }
	}
	
	clock_t start_s = clock();
	cpuMult(a, b, N, c);
	clock_t stop_s = clock();
	std::cout << "Time CPU: " << (stop_s - start_s) / double(CLOCKS_PER_SEC) * 1000 <<  " ms\n";
	
	return 0;
}